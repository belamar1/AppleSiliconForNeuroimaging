#undef DT32
//#define DT32 //<- This should be the ONLY difference between core32 and core64!

#ifdef DT32
 #define flt float
#else
 #define flt double
#endif
/*
https://github.com/sol-prog/cuda_cublas_curand_thrust
https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-curand-thrust/

nvcc cuda_mmul.cu -lcublas -lcurand -o cuda_mmul; ./cuda_mmul
matrix multiplication 10 repetitions 32-bit
mmul: min/mean	2367	2375	ms
mmulCUDA: min/mean	22	32	ms
7772416 values, differences 63.0876%, max difference 8.38861e+06
mmul>nvcc cuda_mmul.cu -lcublas -lcurand -o cuda_mmul; ./cuda_mmul
matrix multiplication 10 repetitions 64-bit
mmul: min/mean	2370	2375	ms
mmulCUDA: min/mean	60	70	ms
7772416 values, differences 0%, max difference 0
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>

#ifndef MAX
#define MAX(A,B) ((A) > (B) ? (A) : (B))
#endif

#ifndef MIN
#define MIN(A,B) ((A) > (B) ? (B) : (A))
#endif

double clockMsec() { //return milliseconds since midnight
	struct timespec _t;
	clock_gettime(CLOCK_MONOTONIC, &_t);
	return _t.tv_sec*1000.0 + (_t.tv_nsec/1.0e6);
}

long timediff(double startTimeMsec, double endTimeMsec) {
	return round(endTimeMsec - startTimeMsec);
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(flt *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	#ifdef DT32
	curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
	#else
	hiprandGenerateUniformDouble(prng, A, nr_rows_A * nr_cols_A);
	#endif
}

//naive matrix multiplication, for optimization see http://apfel.mathematik.uni-ulm.de/~lehn/sghpc/gemm/
void mmul(const flt * A, size_t  IA, const flt * B, size_t  IB, flt * C, size_t  IC, size_t M, size_t N, size_t P) {
 /*
    A is regarded as a two-dimensional matrix with dimemnsions [M][P]
    and stride IA.  B is regarded as a two-dimensional matrix with
    dimemnsions [P][N] and stride IB.  C is regarded as a
    two-dimensional matrix with dimemnsions [M][N] and stride IC.

    Pseudocode:     Memory:
    A[m][p]         A[(m*P+p)*IA]
    B[p][n]         B[(p*N+n)*IB]
    C[m][n]         C[(m*N+n)*IC]
These compute:
    for (m = 0; m < M; ++m)
    for (n = 0; n < N; ++n)
        C[m][n] = sum(A[m][p] * B[p][n], 0 <= p < P);
*/

	    for (size_t m = 0; m < M; ++m) {
			size_t mP = m * P;
	    	for (size_t n = 0; n < N; ++n) {
				flt ret = 0.0;
				for (size_t p = 0; p < P; ++p)
	        		ret += A[mP + p] * B[p*N + n];
				C[m*N + n] = ret;
			} //for n
		} //for m
}


// Multiply the arrays A and B on GPU and save the result in C
void gpu_blas_mmul(const flt *A, const flt *B, flt *C, const int m, const int n, const int p) {
	int lda=m,ldb=p,ldc=m;
	const flt alf = 1;
	const flt bet = 0;
	const flt *alpha = &alf;
	const flt *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	#ifdef DT32
	cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, m, n, p, alpha, A, lda, B, ldb, beta, C, ldc);
	#else
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, p, alpha, A, lda, B, ldb, beta, C, ldc);
	
	//cublasDgemm(handle, CUBLAS_OP_T, CUBLAS_OP_T, m, n, p, alpha, A, lda, B, ldb, beta, C, ldc);
	#endif
	// Destroy the handle
	hipblasDestroy(handle);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const flt *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
	size_t m = 485776; //<- voxels
	size_t n = 16; //statistical contrast, e.g "1 0 0" 
	size_t p = 120; //<- shared: participants
	size_t reps = 10;
	printf("matrix multiplication %zu repetitions %llu-bit\n",  reps, (unsigned long long) sizeof(flt)*8);
	
	// Allocate 3 arrays on CPU
	flt *a = (flt *)malloc(m * p * sizeof(flt));
	flt *b = (flt *)malloc(p * n * sizeof(flt));
	flt *cGPU = (flt *)malloc(m * n * sizeof(flt));
	flt *c = (flt *)malloc(m * n * sizeof(flt));
	//
	for (size_t i = 0; i < (m * p); i++)
    		a[i] = (flt)i;//(flt) rand()/RAND_MAX;
	for (size_t i = 0; i < (p* n); i++)
    		b[i] = (flt)i;//(flt) rand()/RAND_MAX;	
 
    //CPU solution:
    long mn = INT_MAX;
	long sum = 0.0;
	for (int64_t i = 0; i < reps; i++) {
		double startTime = clockMsec();
    		mmul(a, 1, b, 1, c, 1, m, n, p);
    		mn = MIN(mn, timediff(startTime, clockMsec()));	
		sum += timediff(startTime, clockMsec());
    }
    printf("mmul: min/mean\t%ld\t%ld\tms\n", mn, sum/reps);
	// Allocate 3 arrays on GPU
	flt *d_A, *d_B, *d_C;
	

	hipMalloc(&d_A, m * p * sizeof(flt));
	hipMalloc(&d_B, p * n * sizeof(flt));
	hipMalloc(&d_C, m * n * sizeof(flt));
    
    mn = INT_MAX;
	sum = 0.0;
	for (int64_t i = 0; i < reps; i++) {
		double startTime = clockMsec();
	// Transfer data to GPU
	hipMemcpy(d_A, a, m * p * sizeof(flt),hipMemcpyHostToDevice);
	hipMemcpy(d_B, b, p * n * sizeof(flt),hipMemcpyHostToDevice);
	
	// Multiply A and B on GPU
	//https://docs.nvidia.com/cuda/cublas/index.html
	// since matrices stored in column-major format
	// we compute "C = B * A" instead of "C = A * B"
	gpu_blas_mmul(d_B, d_A, d_C, n, m, p);
	//gpu_blas_mmul(d_A, d_B, d_C, m, n, p);

	// Copy (and print) the result on host memory
	hipMemcpy(cGPU,d_C,m * n * sizeof(flt),hipMemcpyDeviceToHost);
    		mn = MIN(mn, timediff(startTime, clockMsec()));	
		sum += timediff(startTime, clockMsec());
    }
    printf("mmulCUDA: min/mean\t%ld\t%ld\tms\n", mn, sum/reps);
 
	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	

	//#define dbug
	#ifdef dbug
	std::cout << "A =" << std::endl;
	print_matrix(a, p, m);
	std::cout << "B =" << std::endl;
	print_matrix(b, n, p);
	std::cout << "C(cpu) =" << std::endl;
	print_matrix(c, n, m);
	std::cout << "C(gpu) =" << std::endl;
	print_matrix(cGPU, n, m);

	#endif
	//check results
	size_t nDiff = 0;
    	flt mxDiff = (flt) 0.0;
	for (size_t i = 0; i < (m * n); i++) {
		if (c[i] != cGPU[i]) {
			nDiff ++;
			mxDiff = MAX(mxDiff, fabs(c[i] - cGPU[i]) );
		}
	}
	printf("%zu values, differences %g%%, max difference %g\n", (m * n), ((double) nDiff) / ((double) (m*n)) * 100.0, mxDiff);

	// Free CPU memory
	free(a);
	free(b);
	free(c);
	free(cGPU);
	return 0;
}

